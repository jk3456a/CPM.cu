#include "hip/hip_runtime.h"
/*
 * Modified by Neural Magic
 * Copyright (C) Marlin.2024 Elias Frantar
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include "marlin.cuh"
#include "core/scalar_type.hpp"
#include "gptq_marlin_utils.cuh"
#include "../../utils.cuh"

namespace marlin {

// For a given "a" of size [M,K] performs a permutation of the K columns based
// on the given "perm" indices.
__global__ void permute_cols_kernel(int4 const* __restrict__ a_int4_ptr,
                                    int const* __restrict__ perm_int_ptr,
                                    int4* __restrict__ out_int4_ptr, int size_m,
                                    int size_k, int block_rows) {
  int start_row = block_rows * blockIdx.x;
  int finish_row = start_row + block_rows;
  if (finish_row > size_m) {
    finish_row = size_m;
  }
  int cur_block_rows = finish_row - start_row;

  int row_stride = size_k * sizeof(half) / 16;

  auto permute_row = [&](int row) {
    int iters = size_k / default_threads;
    int rest = size_k % default_threads;

    int offset = row * row_stride;

    half const* a_row_half = reinterpret_cast<half const*>(a_int4_ptr + offset);
    half* out_half = reinterpret_cast<half*>(out_int4_ptr + offset);

    int base_k = 0;

    for (int i = 0; i < iters; i++) {
      int cur_k = base_k + threadIdx.x;
      int src_pos = perm_int_ptr[cur_k];

      out_half[cur_k] = a_row_half[src_pos];

      base_k += default_threads;
    }

    if (rest) {
      if (threadIdx.x < rest) {
        int cur_k = base_k + threadIdx.x;
        int src_pos = perm_int_ptr[cur_k];

        out_half[cur_k] = a_row_half[src_pos];
      }
    }
  };

  for (int i = 0; i < cur_block_rows; i++) {
    int cur_row = start_row + i;
    if (cur_row < size_m) {
      permute_row(cur_row);
    }
  }
}

thread_config_t small_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {128, 128, 256},
    {64, 128, 128},
    {128, 64, 128},
};

thread_config_t large_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {64, 256, 256},
    {64, 128, 128},
    {128, 64, 128},
};

int get_scales_cache_size(thread_config_t const& th_config, int prob_m,
                          int prob_n, int prob_k, int num_bits, int group_size,
                          bool has_act_order, bool is_k_full) {
  bool cache_scales_chunk = has_act_order && !is_k_full;

  int tb_n = th_config.thread_n;
  int tb_k = th_config.thread_k;

  // Get max scale groups per thread-block
  int tb_groups;
  if (group_size == -1) {
    tb_groups = 1;
  } else if (group_size == 0) {
    tb_groups = div_ceil(tb_k, 32);  // Worst case is 32 group size
  } else {
    tb_groups = div_ceil(tb_k, group_size);
  }

  if (cache_scales_chunk) {
    int load_groups =
        tb_groups * pipe_stages * 2;     // Chunk size is 2x pipeline over dim K
    load_groups = max(load_groups, 32);  // We load at least 32 scale groups
    return load_groups * tb_n * 2;

  } else {
    int tb_scales = tb_groups * tb_n * 2;

    return tb_scales * pipe_stages;
  }
}

bool is_valid_cache_size(thread_config_t const& th_config, int max_m_blocks,
                         int prob_m, int prob_n, int prob_k, int num_bits,
                         int scales_cache_size, int max_shared_mem) {
  int pack_factor = 32 / num_bits;

  // Get B size
  int tb_k = th_config.thread_k;
  int tb_n = th_config.thread_n;

  int b_size = (tb_k * tb_n / pack_factor) * 4;

  // Get A size
  int m_blocks = div_ceil(prob_m, 16);
  int tb_max_m = 16;

  while (true) {
    if (m_blocks >= max_m_blocks) {
      tb_max_m *= max_m_blocks;
      break;
    }

    max_m_blocks--;
    if (max_m_blocks == 0) {
      ERROR_CHECK(false, "Unexpected m_blocks = ", m_blocks);
    }
  }

  int a_size = (tb_max_m * tb_k) * 2;

  float pipe_size = (a_size + b_size) * pipe_stages;

  float reduce_size = max(th_config.num_threads * 32 * 4,
                          (tb_n / 64) * 32 * (tb_max_m / 16) * 4 * 2 * 4 * 2);

  ERROR_CHECK(max_shared_mem / 2 > scales_cache_size, "Sanity check failed: max_shared_mem/2 must be > scales_cache_size");  // Sanity

  return pipe_size + reduce_size < 0.95f * (max_shared_mem - scales_cache_size);
}

bool is_valid_config(thread_config_t const& th_config, int max_m_blocks,
                     int prob_m, int prob_n, int prob_k, int num_bits,
                     int group_size, bool has_act_order, bool is_k_full,
                     int max_shared_mem) {
  // Sanity
  if (th_config.thread_k == -1 || th_config.thread_n == -1 ||
      th_config.num_threads == -1) {
    return false;
  }

  // Verify K/N are divisible by thread K/N
  if (prob_k % th_config.thread_k != 0 || prob_n % th_config.thread_n != 0) {
    return false;
  }

  // Verify min for thread K/N
  if (th_config.thread_n < min_thread_n || th_config.thread_k < min_thread_k) {
    return false;
  }

  // num_threads must be at least 128 (= 4 warps)
  if (th_config.num_threads < 128) {
    return false;
  }

  //  Determine cache for scales
  int scales_cache_size =
      get_scales_cache_size(th_config, prob_m, prob_n, prob_k, num_bits,
                            group_size, has_act_order, is_k_full);

  // Check that pipeline fits into cache
  if (!is_valid_cache_size(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                           num_bits, scales_cache_size, max_shared_mem)) {
    return false;
  }

  return true;
}

int determine_reduce_max_m(int prob_m, int max_par) {
  constexpr int tile_m_size = 16;

  if (prob_m <= tile_m_size) {
    return tile_m_size;

  } else if (prob_m <= tile_m_size * 2) {
    return tile_m_size * 2;

  } else if (prob_m <= tile_m_size * 3) {
    return tile_m_size * 3;

  } else if (prob_m <= tile_m_size * 4) {
    return tile_m_size * 4;

  } else {
    int cur_par = min(div_ceil(prob_m, tile_m_size * 4), max_par);
    return tile_m_size * 4 * cur_par;
  }
}

exec_config_t determine_thread_config(int prob_m, int prob_n, int prob_k,
                                      int num_bits, int group_size,
                                      bool has_act_order, bool is_k_full,
                                      int max_shared_mem) {
  int max_m_blocks = 4;
  while (max_m_blocks > 0) {
    if (prob_m <= 16) {
      for (auto th_config : small_batch_thread_configs) {
        if (is_valid_config(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                            num_bits, group_size, has_act_order, is_k_full,
                            max_shared_mem)) {
          return exec_config_t{max_m_blocks, th_config};
        }
      }
    } else {
      for (auto th_config : large_batch_thread_configs) {
        if (is_valid_config(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                            num_bits, group_size, has_act_order, is_k_full,
                            max_shared_mem)) {
          return exec_config_t{max_m_blocks, th_config};
        }
      }
    }

    max_m_blocks--;  // Process less M blocks per invocation to reduce cache
                     // usage
  }

  return exec_config_t{0, {-1, -1, -1}};
}


} // namespace marlin 
